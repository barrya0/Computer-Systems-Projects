#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <ctime>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

int width, height; //image dimensions

//3x3 matrix for convolution
int mask[3][3] = { 1, 2, 1, 
				2, 3, 2, 
				1, 2, 1 };

//get pixel value after applying convolution filter
int Pixelval(unsigned char* arr, int col, int row) {
	//stores convolution result
	int sum = 0;
	for (int j = -1; j <= 1; j++) {
		for (int i = -1; i <= 1; i++) {
			//use mask to get pixel value from array
			int c = arr[(row + j) * width + (col + i)];
			//convolution operation
			sum += c * mask[i + 1][j + 1];
		}
	}
	sum = sum / 15; //normalize the sum
	return sum;
}

//Function that serves the host (CPU) to apply Blur effect sequentially
void hostBlur(unsigned char* arr, unsigned char* result) {
	int offset = 2 * width;
	//excluding 1st and last 3 rows
	for (int row = 2; row < height - 3; row++) {
		//exclude 1st and last 3 columns
		for (int col = 2; col < width - 3; col++) {
			//apply blur effect with Pixelval
			result[offset + col] = Pixelval(arr, col, row);
		}
		offset += width;
	}
}

// Function to load an image using stb_image
void loadImage(const char* filePath, unsigned char** data, int* w, int* h, int* channels) {
	*data = stbi_load(filePath, w, h, channels, 0);
	if (!*data) {
		fprintf(stderr, "Error: Couldn't load the image: %s\n", filePath);
		exit(EXIT_FAILURE);
	}
}

// Function to write an image using stb_image_write
void writeImage(const char* filePath, unsigned int w, unsigned int h, int channels, unsigned char* data) {
	if (!stbi_write_png(filePath, w, h, channels, data, w * channels)) {
		fprintf(stderr, "Error: Couldn't write the image: %s\n", filePath);
		exit(EXIT_FAILURE);
	}
}


// Function to load a PGM file and return image dimensions and pixel data
void loadPGM(const char* filePath, unsigned int* width, unsigned int* height, unsigned char** data) {
	FILE* file = fopen(filePath, "rb");
	if (!file) {
		fprintf(stderr, "Error: Couldn't open the file: %s\n", filePath);
		exit(EXIT_FAILURE);
	}

	// Read PGM header
	char magic[3];
	fscanf(file, "%2s", magic);
	if (strcmp(magic, "P5") != 0) {
		fprintf(stderr, "Error: Not a PGM file (magic number P5 expected)\n");
		exit(EXIT_FAILURE);
	}

	fscanf(file, "%u %u", width, height);
	fgetc(file); // Ignore one whitespace character
	unsigned int maxValue;
	fscanf(file, "%u", &maxValue);

	// Allocate memory for pixel data
	*data = (unsigned char*)malloc((*width) * (*height));

	// Read pixel data
	fread(*data, sizeof(unsigned char), (*width) * (*height), file);

	fclose(file);
}

// Function to write PGM image to a file
void writePGM(const char* filePath, unsigned int width, unsigned int height, unsigned char* data) {
	FILE* file = fopen(filePath, "wb");
	if (!file) {
		fprintf(stderr, "Error: Couldn't open the file for writing: %s\n", filePath);
		exit(EXIT_FAILURE);
	}

	// Write PGM header
	fprintf(file, "P5\n%d %d\n255\n", width, height);

	// Write pixel data
	fwrite(data, sizeof(unsigned char), width * height, file);

	fclose(file);
}

//Function that serves the device (GPU) to apply Blur effect with parallelism
__global__ void deviceBlur(unsigned char* arr, unsigned char* result, int width, int height) {
	//Calculate column and row indices for each thread
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	//Is current thread outside of valid image area?
	if (row < 2 || col < 2 || row >= height - 3 || col >= width - 3)
		return;

	//Same process as seen in Pixelval function
	//define blur mask within kernel
	int mask[3][3] = {
		{1,2,1},
		{2,3,2},
		{1,2,1}
	};
	int sum = 0;
	for (int j = -1; j <= 1; j++) {
		for (int i = -1; i <= 1; i++) {
			//use mask to get pixel value from array
			int c = arr[(row + j) * width + (col + i)];
			//convolution operation
			sum += c * mask[i + 1][j + 1];
		}
	}
	result[row * width + col] = sum / 15;
}

// Function to apply blur using CUDA
void blurUsingCUDA(unsigned char* d_arr, unsigned char* d_result, unsigned int width, unsigned int height) {
	// Set up the grid and block dimensions
	dim3 threadsPerBlock(16, 16);
	//dim3 numBlocks(width/16, height/16); may work better only with PGMS
	dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

	// Launch the CUDA kernel
	deviceBlur<<<numBlocks, threadsPerBlock>>>(d_arr, d_result, width, height);

	// Ensure all threads have finished
	hipDeviceSynchronize();
}

int main(int argc, char** argv) {
	unsigned char* h_res;
	unsigned char* d_res;
	unsigned char* hPix = NULL;
	unsigned char* dPix = NULL;

	const char* image = "pathToYourImage";
	const char* hostResultPath = "resultPath";
	const char* deviceResultPath = "resultPath";

	//populate data from pgm onto host
	//loadPGM(image, &width, &height, &hPix);
	int channels;
	hPix = stbi_load(image, &width, &height, &channels, 0);
	if (!hPix) {
		fprintf(stderr, "Error: Couldn't load the image: %s\n", image);
		exit(EXIT_FAILURE);
	}

	int imageSize = sizeof(unsigned char) * width * height * channels;

	h_res = (unsigned char*)malloc(imageSize);
	hipMalloc((void**)&dPix, imageSize);
	hipMalloc((void**)&d_res, imageSize);
	hipMemcpy(dPix, hPix, imageSize, hipMemcpyHostToDevice);

	clock_t start, end, elapsed;
	start = clock();

	hostBlur(hPix, h_res);
	end = clock();
	elapsed = (end - start);
	double diff = elapsed / (double)CLOCKS_PER_SEC;
	printf("CPU time = %.2f ms\n", diff * 1000);

	//writePGM(hostResultPath, width, height, h_res);
	writeImage(hostResultPath, width, height, channels, h_res);

	start = clock();
	//Invoke the kernel
	blurUsingCUDA(dPix, d_res, width, height);
	end = clock();
	elapsed = (end - start);
	diff = elapsed / (double)CLOCKS_PER_SEC;
	printf("GPU time = %.2f ms\n", diff * 1000);

	hipMemcpy(h_res, d_res, imageSize, hipMemcpyDeviceToHost);

	//writePGM(deviceResultPath, width, height, h_res);
	writeImage(deviceResultPath, width, height, channels, h_res);

	//Free allocated memory
	hipFree(dPix);
	hipFree(d_res);
	stbi_image_free(hPix);
	free(h_res);
	return 0;
}
